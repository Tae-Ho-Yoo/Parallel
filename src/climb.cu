#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "utils.h"

#define N_THREADS 16
#define N_BLOCKS 1024

/*** GPU functions ***/
__global__ void init_rand_kernel(hiprandState *state) {
 int idx = blockIdx.x * blockDim.x + threadIdx.x;
 hiprand_init(0, idx, 0, &state[idx]);
}

__global__ void random_walk_kernel(float *map, int rows, int cols, int* bx, int* by,
                                   int steps, hiprandState *state) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  float randval = hiprand_uniform(&state[tid]);
  int randint = int(randval * (rows * cols));
  int curridx = randint;

  float max_height = map[curridx];

  bx[tid] = curridx % rows;
  by[tid] = curridx / rows;

  for (int i = 0; i < steps; i++){
    int randact = int(randval * 4);

    if (randact == 0){
      if (curridx + 1 < rows * cols){
        curridx++;
      }
    }
    else if (randact == 1){
      if (curridx - 1 > 0){
        curridx--;
      }
    }
    else if (randact == 2){
      if (curridx + rows < rows * cols){
        curridx += rows;
      }
    }
    else if (randact == 3){
      if (curridx - rows > 0){
        curridx -= rows;
      }
    }

    if (map[curridx] > max_height){
      max_height = map[curridx];
      bx[tid] = curridx % rows;
      by[tid] = curridx / rows;
    }
  }
}

__global__ void local_max_kernel(float *map, int rows, int cols, int* bx, int* by,
                                 int steps, hiprandState *state) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  //TODO: implement local max!
}

__global__ void local_max_restart_kernel(float *map, int rows, int cols, int* bx,
                                         int* by, int steps, hiprandState *state) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  //TODO: implement local max with restarts!
}

// /*** CPU functions ***/
// hiprandState* init_rand() {
//   hiprandState *d_state;
//   hipMalloc(&d_state, N_BLOCKS * N_THREADS * sizeof(hiprandState));
//   init_rand_kernel<<<N_BLOCKS, N_THREADS>>>(d_state);
//   return d_state;
// }

float random_walk(float* map, int rows, int cols, int steps) {
  hiprandState* d_state;
  int *bx, *by;
  int *d_bx, *d_by;
  float* d_map;

  bx = (int *)malloc(N_BLOCKS * N_THREADS  * sizeof(int));
  by = (int *)malloc(N_BLOCKS * N_THREADS  * sizeof(int));

  for (int i = 0; i < N_BLOCKS * N_THREADS; i++){
    bx[i] = i;
    by[i] = i;
  }

  hipMalloc(&d_state, N_BLOCKS * N_THREADS * sizeof(hiprandState));
  hipMalloc(&d_bx, N_BLOCKS * N_THREADS  * sizeof(int));
  hipMalloc(&d_by, N_BLOCKS * N_THREADS  * sizeof(int));
  hipMalloc(&d_map, rows * cols * sizeof(int));

  hipMemcpy(d_bx, bx, N_BLOCKS * N_THREADS  * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_by, by, N_BLOCKS * N_THREADS  * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_map, map, rows * cols * sizeof(float), hipMemcpyHostToDevice);

  // Before kernel call:
  // Need to allocate memory for above variables and copy data to GPU
  init_rand_kernel<<<N_BLOCKS, N_THREADS>>>(d_state);
  random_walk_kernel<<<N_BLOCKS, N_THREADS>>>(d_map, rows, cols, d_bx, d_by, steps, d_state);

  // After kernel call:
  // Need to copy data back to CPU and find max value

  hipMemcpy(bx, d_bx, N_BLOCKS * N_THREADS  * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(by, d_by, N_BLOCKS * N_THREADS  * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(map, d_map, rows * cols * sizeof(float), hipMemcpyDeviceToHost);

  float max_val = 0;

  for (int i = 0; i < N_BLOCKS * N_THREADS; i++){
    for (int j = 0; j < N_BLOCKS * N_THREADS; j++){
      if (map[rows * by[i] + bx[j]] > max_val){
        max_val = map[rows * by[i] + bx[j]];
        printf("%f\n", max_val);
      }
    }
  }
  
  // Finally: free used GPU and CPU memory

  hipFree(d_bx);
  hipFree(d_by);
  hipFree(d_map);
  hipFree(d_state);

  free(bx);
  free(by);
  free(map);

  return max_val;
}

// Work on these after finishing random walk
float local_max(float* map, int rows, int cols, int steps);
float local_max_restart(float* map, int rows, int cols, int steps);


int main(int argc, char** argv) {
  if (argc != 2) {
    printf("Usage: %s <map_file> \n", argv[0]);
    return 1;
  }

  float *map;
  int rows, cols;
  read_bin(argv[1], &map, &rows, &cols);

  printf("%d %d\n", rows, cols);

  // As a starting point, try to get it working with a single steps value
  int steps = 1;
  float max_val = random_walk(map, rows, cols, steps);
  printf("Random walk max value: %f\n", max_val);

  return 0;
}
